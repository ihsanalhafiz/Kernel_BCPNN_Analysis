#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <kernelGlobal.cuh>
#include <hip/hip_math_constants.h>
#include "prj.cuh"


__global__
void updtrcjzp_kernel_optimized(const float* __restrict__ Xj,
                                int Nj,
                                const float fgain, const float eps,
                                const float tauzjdt, const float taupdt,
                                float* __restrict__ Zj,
                                float* __restrict__ Pj) {
    int nj = blockIdx.x * blockDim.x + threadIdx.x;
    if (nj >= Nj)
        return;

    // Load data into registers
    float Xj_nj = Xj[nj];
    float Zj_nj = Zj[nj];
    float Pj_nj = Pj[nj];

    // Compute delta Zj
    float delta_Zj = (fgain * Xj_nj * (1.0f - eps) + eps - Zj_nj) * tauzjdt;
    Zj_nj += delta_Zj;

    // Update Zj and Pj
    Zj[nj] = Zj_nj;
    Pj[nj] = Pj_nj + (Zj_nj - Pj_nj) * taupdt;
}

__global__
void updtrcizp_kernel_optimized(const float *__restrict__ Xi,
                      int total_elements,
                      float fgain, float eps, float tauzidt, float taupdt,
                      float *__restrict__ Zi, float *__restrict__ Pi) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= total_elements)
        return;

    // Load data into registers
    float Xi_k = Xi[k];
    float Zi_k = Zi[k];
    float Pi_k = Pi[k];

    // Compute delta Zi
    float delta_Zi = (Xi_k * fgain * (1.0f - eps) + eps - Zi_k) * tauzidt;
    Zi_k += delta_Zi;

    // Update Zi and Pi
    Zi[k] = Zi_k;
    Pi[k] = Pi_k + (Zi_k - Pi_k) * taupdt;
}

__global__
void updtrcjip_kernel_optimized(const float* __restrict__ Zj,
                                const float* __restrict__ Zi,
                                int Nj, int Mj, int denNi,
                                const float taupdt,
                                float* __restrict__ Pji) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= Nj * denNi)
        return;

    int nj = n / denNi;
    int ni = n % denNi;
    int hj = nj / Mj;
    int Zi_idx = hj * denNi + ni;
    int Pji_idx = nj * denNi + ni;

    // Compute delta Pji
    float delta_Pji = (Zi[Zi_idx] * Zj[nj] - Pji[Pji_idx]) * taupdt;

    // Update Pji
    Pji[Pji_idx] += delta_Pji;
}

void updtraces_cu_optimized(const float* __restrict__ denact,
                  const float* __restrict__ trgact,
                  float prn,
                  int Hj, int Nj, int Mj, int denNi,
                  float fgain, float eps,
                  float tauzidt, float tauzjdt, float taupdt,
                  float* __restrict__ Zj,
                  float* __restrict__ Zi,
                  float* __restrict__ Pj,
                  float* __restrict__ Pi,
                  float* __restrict__ Pji) {
    float prntaupdt = prn * taupdt;
    int blockSize_1 = 256;  // Adjusted block size for better occupancy
    int blockSize = 128;  // Adjusted block size for better occupancy

    // Kernel 1: updtrcjzp_kernel_optimized
    int numBlocksj = (Nj + blockSize_1 - 1) / blockSize_1;
    updtrcjzp_kernel_optimized<<<numBlocksj, blockSize_1>>>(
        trgact, Nj, fgain, eps, tauzjdt, prntaupdt, Zj, Pj);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in updtrcjzp_kernel_optimized: %s\n", hipGetErrorString(err));
    }

    // Kernel 2: updtrcizp_kernel_optimized
    int total_elements_i = Hj * denNi;
    int numBlocksi = (total_elements_i + blockSize_1 - 1) / blockSize_1;
    updtrcizp_kernel_optimized<<<numBlocksi, blockSize_1>>>(denact, total_elements_i, fgain, eps, tauzidt, prntaupdt, Zi, Pi);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in updtrcizp_kernel_optimized: %s\n", hipGetErrorString(err));
    }

    // Kernel 3: updtrcjip_kernel_optimized
    int total_elements_ji = Nj * denNi;
    int numBlocksji = (total_elements_ji + blockSize - 1) / blockSize;
    updtrcjip_kernel_optimized<<<numBlocksji, blockSize>>>(
        Zj, Zi, Nj, Mj, denNi, prntaupdt, Pji);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in updtrcjip_kernel_optimized: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();
}

__global__
void compute_Bj_kernel(int Nj,
                       const float* __restrict__ Pj,
                       float* __restrict__ Bj,
                       const float bgain) {
    int nj = blockIdx.x * blockDim.x + threadIdx.x;
    if (nj >= Nj)
        return;

    // Compute Bj[nj]
    Bj[nj] = bgain * logf(Pj[nj]);
}

__global__
void BCPupdbw_kernel_optimized(int Nj, int Mj, int denHi, int denNi, int Mi,
                               const float* __restrict__ Pj,
                               const float* __restrict__ Pi,
                               const float* __restrict__ Pji,
                               const float* __restrict__ Bj,
                               float* __restrict__ Wji,
                               const float eps,
                               const float wgain,
                               const float ewgain,
                               const float iwgain) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = Nj * denNi;

    if (idx >= total_elements)
        return;

    int nj = idx / denNi;
    int dni = idx % denNi;
    int hj = nj / Mj;
    int k = hj * denNi + dni;

    // Load values into registers
    float Pj_nj = Pj[nj];
    float Pi_k = Pi[k];
    float Pji_idx = Pji[idx];

    // Compute wji
    float wji = logf(Pji_idx / (Pi_k * Pj_nj));

    // Compute gain without branch divergence
    float pos_mask = (wji > 0.0f);
    float neg_mask = (wji < 0.0f);
    float gain = wgain + ewgain * pos_mask + iwgain * neg_mask;

    // Update wji
    wji *= gain;

    // Write result to global memory
    Wji[idx] = wji;
}

void updbw_cu_optimized(int Nj, int Mj, int denHi, int denNi, int Mi,
              const float* Pj,
              const float* Pi,
              const float* Pji,
              float* Bj,
              float* Wji,
              float eps,
              float bgain,
              float wgain,
              float ewgain,
              float iwgain) {
    // Compute Bj[nj] separately to avoid redundant computations
    int blockSizeBj = 256;
    int numBlocksBj = (Nj + blockSizeBj - 1) / blockSizeBj;

    compute_Bj_kernel<<<numBlocksBj, blockSizeBj>>>(
        Nj, Pj, Bj, bgain);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in compute_Bj_kernel: %s\n", hipGetErrorString(err));
    }

    // Synchronize before launching the next kernel
    hipDeviceSynchronize();

    // Launch the optimized BCPupdbw_kernel
    int total_elements = Nj * denNi;
    int blockSize = 128;  // Adjust based on GPU occupancy
    int numBlocks = (total_elements + blockSize - 1) / blockSize;

    BCPupdbw_kernel_optimized<<<numBlocks, blockSize>>>(
        Nj, Mj, denHi, denNi, Mi,
        Pj, Pi, Pji, Bj, Wji,
        eps, wgain, ewgain, iwgain);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error in BCPupdbw_kernel_optimized: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();
}

__global__
void updbwsup_kernel_optimized(const float* __restrict__ bwsupinf,
                     int Nj,
                     const float tauzidt,
                     float* __restrict__ bwsup) {
    int nj = blockIdx.x * blockDim.x + threadIdx.x;
    if (nj >= Nj)
        return;

    float bwsup_nj = bwsup[nj];
    float bwsupinf_nj = bwsupinf[nj];
    bwsup_nj += (bwsupinf_nj - bwsup_nj) * tauzidt;
    bwsup[nj] = bwsup_nj;
}

void updbwsup_cu_optimized(const float* __restrict__ Zi,
                 const float* __restrict__ Bj,
                 const float* __restrict__ Wji,
                 int Hj, int Mj, int denNi, float tauzidt,
                 float* __restrict__ bwsupinf,
                 float* __restrict__ bwsup) {
    int Nj = Hj * Mj;
    float alpha = 1.0f, beta = 0.0f;

    // Initialize cuBLAS handle
    hipblasHandle_t handle;
    CUBLAS_CHECK_ERROR(hipblasCreate(&handle));

    // Allocate host arrays of device pointers
    float** h_Aarray = (float**)malloc(Hj * sizeof(float*));
    float** h_xarray = (float**)malloc(Hj * sizeof(float*));
    float** h_yarray = (float**)malloc(Hj * sizeof(float*));

    // Initialize host arrays with device pointers
    for (int hj = 0; hj < Hj; hj++) {
        h_Aarray[hj] = (float*)(Wji + hj * Mj * denNi);
        h_xarray[hj] = (float*)(Zi + hj * denNi);
        h_yarray[hj] = (float*)(bwsupinf + hj * Mj);
    }

    // Allocate device arrays of pointers
    float** d_Aarray;
    float** d_xarray;
    float** d_yarray;
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_Aarray, Hj * sizeof(float*)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_xarray, Hj * sizeof(float*)));
    CUDA_CHECK_ERROR(hipMalloc((void**)&d_yarray, Hj * sizeof(float*)));

    // Copy host arrays to device arrays
    CUDA_CHECK_ERROR(hipMemcpy(d_Aarray, h_Aarray, Hj * sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(d_xarray, h_xarray, Hj * sizeof(float*), hipMemcpyHostToDevice));
    CUDA_CHECK_ERROR(hipMemcpy(d_yarray, h_yarray, Hj * sizeof(float*), hipMemcpyHostToDevice));

    // Call hipblasSgemvBatched
    CUBLAS_CHECK_ERROR(hipblasSgemvBatched(
        handle,
        HIPBLAS_OP_T,         // Transpose operation
        denNi,               // m
        Mj,                  // n
        &alpha,
        (const float**)d_Aarray, denNi,  // Aarray and leading dimension
        (const float**)d_xarray, 1,      // xarray and increment
        &beta,
        d_yarray, 1,                     // yarray and increment
        Hj));                            // Batch count

    // Free device arrays of pointers
    CUDA_CHECK_ERROR(hipFree(d_Aarray));
    CUDA_CHECK_ERROR(hipFree(d_xarray));
    CUDA_CHECK_ERROR(hipFree(d_yarray));

    // Free host arrays of pointers
    free(h_Aarray);
    free(h_xarray);
    free(h_yarray);

    // Launch the kernel to update bwsup using the GPU
    int blockSize = 256;  // Adjust for better occupancy
    int numBlocksj = (Nj + blockSize - 1) / blockSize;
    updbwsup_kernel_optimized<<<numBlocksj, blockSize>>>(
        bwsupinf, Nj, tauzidt, bwsup);

    // Check for CUDA kernel launch errors and synchronize
    CUDA_CHECK_ERROR(hipPeekAtLastError());
    CUDA_CHECK_ERROR(hipDeviceSynchronize());

    // Destroy cuBLAS handle
    CUBLAS_CHECK_ERROR(hipblasDestroy(handle));
}
